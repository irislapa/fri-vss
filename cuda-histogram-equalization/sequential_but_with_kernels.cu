#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "include/hip/hip_runtime_api.h"

#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "include/stb_image.h"
#include "include/stb_image_write.h"

#define GRAYLEVELS 256
#define COLOR_CHANNELS 1
#define DESIRED_NCHANNELS 1

#define N 1

#define PRINT_HISTOGRAMS

//#define CDF_NAIVE
#define CDF_WE
//#define CDF_WE_MBCDF

#ifdef CDF_WE_MBCF 
    #define NUM_BANKS 16
    #define LOG_NUM_BANKS 4
    #define CONFLICT_FREE_OFFSET(n) (((n) >> NUM_BANKS) + ((n) >> (2 * LOG_NUM_BANKS)))
#endif  // PARALLEL_CDF_WE_MBCF

__global__ void findMinKernel(unsigned int* cdf, unsigned int*d_cdfmin) {
    if (threadIdx.x == 0) {
        unsigned int min = 0;
        // grem skozi CDF dokler ne najdem prvi nenicelni element ali pridem do konca
        for (int i = 0; min == 0 && i < GRAYLEVELS; i++) {
		    min = cdf[i];
        }
    
        *d_cdfmin = min;
    }



}

/**************1st step: CALCULATE HISTOGRAM ****************/

/*************** KERNEL FOR CALCULATING HISTOGRAM "LOCALLY" ***************/
// each block (16*16)threads, calculates its local histogram
// then the local histograms are summed to get the global histogram
__global__ void CalculateHistogramKernel(unsigned char* image, int width, int height, unsigned int *histogram){
   if (threadIdx.x == 0) {
        for (int i=0; i<height; i++) {
            for (int j=0; j<width; j++) {
                histogram[image[i*width + j]]++;
            }
        }
    }
}



#ifdef CDF_WE
/*************** WORK EFFICIENT KERNEL FOR PARALLEL CDF CLALCULATION ***************/
// code for work efficient parallel cdf, based on the following source:
// https://developer.nvidia.com/gpugems/gpugems3/part-vi-gpu-computing/chapter-39-parallel-prefix-sum-scan-cuda
__global__ void CalculateCDF_we(unsigned int* histogram, unsigned int*cdf) {
    if (threadIdx.x == 0) {
        // clear cdf:
        for (int i=0; i<GRAYLEVELS; i++) {
            cdf[i] = 0;
        }
        // calculate cdf from histogram
        cdf[0] = histogram[0];
        for (int i=1; i<GRAYLEVELS; i++) {
            cdf[i] = cdf[i-1] + histogram[i];
        }
    }
}
#endif  // CDF_WE





__device__ unsigned char scale(unsigned int cdf, unsigned int cdfmin, unsigned int imageSize) {
    float scale;
    scale = (float)(cdf - cdfmin) / (float)(imageSize - cdfmin);
    scale = round(scale * (float)(GRAYLEVELS-1));
    return (int)scale;
}


/**************3rd step: EQUALIZE ****************/
__global__ void EqualizeKernel(unsigned char * image_in, unsigned char * image_out, int width, int height, unsigned int *cdf, unsigned int *cdfmin) {
    if (threadIdx.x == 0) {
        unsigned int imageSize = width * height;
        //Equalize: namig: blok niti naj si CDF naloži v skupni pomnilnik
        for (int i=0; i<height; i++) {
            for (int j=0; j<width; j++) {
                image_out[(i*width + j)] = scale(cdf[image_in[i*width + j]], *cdfmin, imageSize);
            }
        }
    }
}



int main(int argc, char *argv[]) {
    
    char imageInName[255]; char imageoutNamePNG[255]; char imageoutNameJPG[255]; char imageInFormat[5];
    snprintf(imageInName, 255, "%s", argv[1]);
    snprintf(imageInFormat, 5, "%s", argv[2]); 
    strncat(imageInName, imageInFormat, 4);
    snprintf(imageoutNamePNG, 255, "%s", argv[1]); snprintf(imageoutNameJPG, 255, "%s", argv[1]);
    strncat(imageoutNamePNG, "_out.png", 12); strncat(imageoutNameJPG, "_out.jpg", 12);

    // Read image from file
    int width, height, cpp;
    // read only DESIRED_NCHANNELS channels from the input image:
    unsigned char *h_imageIn = stbi_load(imageInName, &width, &height, &cpp, DESIRED_NCHANNELS);
    if(h_imageIn == NULL) {
        printf("Error in loading the image\n");
        return 1;
    }
    printf("Loaded image W = %d, H = %d, actual cpp = %d \n", width, height, cpp);
	
    //Allocate memory for raw output image data, histogram, and CDF 
    unsigned char *h_imageOut = (unsigned char *)malloc(width * height * sizeof(unsigned char));
	unsigned int *h_histogram = (unsigned int *)malloc(GRAYLEVELS * sizeof(unsigned int));
    unsigned int *h_cdf = (unsigned int *)malloc(GRAYLEVELS * sizeof(unsigned int));
    unsigned int *h_cdfmin = (unsigned int *)malloc(sizeof(unsigned int));
    

    dim3 blockSize(16, 16);
    dim3 gridSize(ceil((float) width / blockSize.x), ceil((float) height / blockSize.y));
    unsigned int *d_histogram;
    unsigned char *d_imageIn;
    unsigned char *d_imageOut;
    unsigned int *d_cdf;
    unsigned int *d_cdfmin;
    hipMalloc(&d_histogram, GRAYLEVELS * sizeof(unsigned int));
    hipMalloc(&d_imageIn, width * height * sizeof(unsigned char));
    hipMalloc(&d_imageOut, width * height * sizeof(unsigned char));
    hipMalloc(&d_cdf, GRAYLEVELS * sizeof(unsigned int));
    hipMalloc(&d_cdfmin, sizeof(unsigned int));

    //timestart
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    float milliseconds = 0;

	// Histogram equalization steps: 
	// 1. Create the histogram for the input grayscale image.
    hipMemcpy(d_imageIn, h_imageIn, width * height * sizeof(unsigned char), hipMemcpyHostToDevice);
    
    for (int n = 0; n < N; n++){

        CalculateHistogramKernel<<<1, 1>>>(d_imageIn, width, height, d_histogram);
       
        

        #ifdef CDF_WE
        CalculateCDF_we<<<1, 1>>>(d_histogram, d_cdf);
        #endif  // CDF_WE

       
    
	    //  3. Calculate the new gray-level values through the general histogram equalization formula and assign new pixel values
        findMinKernel<<<1, 1>>>(d_cdf, d_cdfmin);
        hipMemcpy(h_cdfmin, d_cdfmin, sizeof(unsigned int), hipMemcpyDeviceToHost);
        //printf("CDFMIN: %d\n", *h_cdfmin);
        //hipMemcpy(h_cdf, d_cdf, GRAYLEVELS * sizeof(unsigned int), hipMemcpyDeviceToHost);

        EqualizeKernel<<<gridSize, blockSize>>>(d_imageIn, d_imageOut, width, height, d_cdf, d_cdfmin);
        //Equalize(h_imageIn, h_imageOut, width, height, h_cdf);
        hipMemcpy(h_imageOut, d_imageOut, height * width * sizeof(unsigned char), hipMemcpyDeviceToHost);
    }

    //time stop
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Kernel Execution time is: %0.4f milliseconds \n", milliseconds/N);


    //Free cuda memory
    hipFree(d_histogram);
    hipFree(d_imageIn);
    hipFree(d_imageOut);
    hipFree(d_cdf);
    hipFree(d_cdfmin);

    // write output image:
    stbi_write_png(imageoutNamePNG, width, height, DESIRED_NCHANNELS, h_imageOut, width * DESIRED_NCHANNELS);
    stbi_write_jpg(imageoutNameJPG, width, height, DESIRED_NCHANNELS, h_imageOut, 100);
    //stbi_write_jpg("out.jpg", width, height, DESIRED_NCHANNELS, h_imageOut, 100);

	//Free memory
	free(h_imageIn);
    free(h_imageOut);
	free(h_histogram);
    free(h_cdf);

    

	return 0;
}

