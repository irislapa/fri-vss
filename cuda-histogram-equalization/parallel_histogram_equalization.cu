#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "include/hip/hip_runtime_api.h"

#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "include/stb_image.h"
#include "include/stb_image_write.h"

#define GRAYLEVELS 256
#define COLOR_CHANNELS 1
#define DESIRED_NCHANNELS 1

#define N 1 

//#define PRINT_HISTOGRAMS

//#define CDF_NAIVE
#define CDF_WE
//#define CDF_WE_MBCDF

#ifdef CDF_WE_MBCF 
    #define NUM_BANKS 16
    #define LOG_NUM_BANKS 4
    #define CONFLICT_FREE_OFFSET(n) (((n) >> NUM_BANKS) + ((n) >> (2 * LOG_NUM_BANKS)))
#endif  // PARALLEL_CDF_WE_MBCF

__global__ void findMinKernel(unsigned int* cdf, unsigned int*d_cdfmin) {
    // Allocate shared memory
	__shared__ unsigned int partial_mins[256];

	// Calculate thread ID
	int tid = threadIdx.x;
    // Load elements into shared memory
    // we are looking for the smallest NON-ZERO value in CDF so we can UINT_MAX all the zeros
    if (tid < 128) {
	    partial_mins[tid] = cdf[tid] == 0 ? UINT_MAX : cdf[tid];
        partial_mins[tid + 128] = cdf[tid + 128] == 0 ? UINT_MAX : cdf[tid + 128];
    }   
    // Start at 1/2 block stride and divide by two each iteration
	for (int s = GRAYLEVELS/2; s > 0; s >>= 1) {
        __syncthreads();
		// Each thread does work unless it is further than the stride
		if (tid < s) {
		    partial_mins[tid] = min(partial_mins[tid], partial_mins[tid + s]); 
	    }
	}
    __syncthreads();
	if (threadIdx.x == 0) {
        *d_cdfmin = partial_mins[0];
	}
}




/**************1st step: CALCULATE HISTOGRAM ****************/

/*************** KERNEL FOR CALCULATING HISTOGRAM "LOCALLY" ***************/
// each block (16*16)threads, calculates its local histogram
// then the local histograms are summed to get the global histogram
__global__ void CalculateHistogramKernel(unsigned char* image, int width, int height, unsigned int *histogram){
   
    // calculate global x, y of pixel on image
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // calculate local x, y of pixel in block 
    int lx = threadIdx.x;
    int ly = threadIdx.y;

    // allocate local memory for local histogram for each block
    __shared__ unsigned int localHistogram[GRAYLEVELS];

    // each thread sets its pixel in local histogram to 0 
    localHistogram[blockDim.x * ly + lx] = 0;
    __syncthreads();

    //read value from image and increment local histogram
    if (x < width && y < height) {
        atomicAdd(&(localHistogram[image[y * width + x]]), 1);
    }
    __syncthreads();
    // now we have a calculation of local histogram

    // now each threadd takes its own beam to global memory, because neighbour threads take neighbour beams to
    // neighbouring memory locations in global memory, we can combine memory accesses (memory coalescing)
    atomicAdd(&(histogram[ly * blockDim.x + lx]), localHistogram[ly * blockDim.x + lx]);
}

#ifdef CDF_NAIVE
/*************** NAIVE KERNEL FOR PARALLEL CDF CLALCULATION ***************/
__global__ void CalculateCDF_naive(unsigned int* histogram, unsigned int *cdf) {
    __shared__ unsigned int temp[GRAYLEVELS*2];
    int tid = threadIdx.x;

    int pout = 0, pin = 1;

    temp[tid] = histogram[tid];

    __syncthreads();

    for(int offset = 1; offset < GRAYLEVELS; offset <<= 1) {
        pout = 1 - pout;
        pin = 1 - pout;
        if (tid >= offset) {
            temp[pout*GRAYLEVELS + tid] = temp[pin*GRAYLEVELS + tid] + temp[pin*GRAYLEVELS + tid - offset];
        } else {
            temp[pout*GRAYLEVELS + tid] = temp[pin*GRAYLEVELS + tid];
        }
        __syncthreads();
    }
    cdf[tid] = temp[pout*GRAYLEVELS + tid];
}
#endif  // CDF_NAIVE

#ifdef CDF_WE
/*************** WORK EFFICIENT KERNEL FOR PARALLEL CDF CLALCULATION ***************/
// code for work efficient parallel cdf, based on the following source:
// https://developer.nvidia.com/gpugems/gpugems3/part-vi-gpu-computing/chapter-39-parallel-prefix-sum-scan-cuda
__global__ void CalculateCDF_we(unsigned int* histogram, unsigned int*cdf) {
    __shared__ unsigned int temp[GRAYLEVELS];

    int tid = threadIdx.x; // 1block 1x128 threads, 128 threads 
    int offset = 1; // distance between elements in array that will be summed

    // the sum of values, that each thredad calculates in 1st step
    temp[2*tid] = histogram[2*tid];
    temp[2*tid+1] = histogram[2*tid+1];


    for (int d = GRAYLEVELS >> 1; d > 0; d >>= 1) {
        __syncthreads();

        if (tid < d) {
            int ai = offset*(2*tid+1)-1;
            int bi = offset*(2*tid+2)-1;
            temp[bi] += temp[ai];
        }
        offset *= 2;
    }

    if (tid == 0) {
        temp[GRAYLEVELS - 1] = 0;
    }
    
    for (int d = 1; d < GRAYLEVELS; d *= 2) {
        offset >>= 1;
        __syncthreads();

        if (tid < d) {
            int ai = offset*(2*tid+1)-1;
            int bi = offset*(2*tid+2)-1;

            float t = temp[ai];
            temp[ai] = temp[bi];
            temp[bi] += t;
        }
    }
    __syncthreads();
    cdf[2*tid] = temp[2*tid];
    cdf[2*tid+1] = temp[2*tid+1];
}
#endif  // CDF_WE

#ifdef CDF_WE_MBCF
/*************** WORK EFFICIENT KERNEL FOR PARALLEL CDF CLALCULATION WITHOUT MEMORY BANK COFNILCTS***************/
/* 
    need to figure out, why calculated cdf values are higher than they should be,
    outputed image still seems to be equalized correctly
*/ 
__global__ void CalculateCDF_we_mbcf(unsigned int* histogram, unsigned int *cdf) {
    __shared__ unsigned int temp[GRAYLEVELS + CONFLICT_FREE_OFFSET(GRAYLEVELS)];

    int tid = threadIdx.x; // 1block 1x128 threads, 128 threads 
    int offset = 1; // distance between elements in array that will be summed

    int ai= 2*tid;
    int bi= tid + (GRAYLEVELS/2);
    int bankOffsetA = CONFLICT_FREE_OFFSET(ai);
    int bankOffsetB = CONFLICT_FREE_OFFSET(bi);
    
    temp[ai + bankOffsetA] = histogram[ai];
    temp[bi + bankOffsetB] = histogram[bi];

    // the sum of values, that each thredad calculates in 1st step

    for (int d = GRAYLEVELS >> 1; d > 0; d >>= 1) {
        __syncthreads();

        if (tid < d) {
            ai = offset*(2*tid+1)-1;
            bi = offset*(2*tid+2)-1;
            temp[bi] += temp[ai];
        }
        offset *= 2;
    }

    if (tid == 0) {
        temp[GRAYLEVELS - 1 + CONFLICT_FREE_OFFSET(GRAYLEVELS - 1)] = 0;
    }
    
    for (int d = 1; d < GRAYLEVELS; d *= 2) {
        offset >>= 1;
        __syncthreads();

        if (tid < d) {
            ai = offset*(2*tid+1)-1;
            bi = offset*(2*tid+2)-1;

            float t = temp[ai];
            temp[ai] = temp[bi];
            temp[bi] += t;
        }
    }
    __syncthreads();
    cdf[ai] = temp[ai + bankOffsetA];
    cdf[bi] = temp[bi + bankOffsetB];
}
#endif  // CDF_WE_MBCF



__device__ unsigned char scale(unsigned int cdf, unsigned int cdfmin, unsigned int imageSize) {
    float scale;
    scale = (float)(cdf - cdfmin) / (float)(imageSize - cdfmin);
    scale = round(scale * (float)(GRAYLEVELS-1));
    return (int)scale;
}


/**************3rd step: EQUALIZE ****************/
__global__ void EqualizeKernel(unsigned char * image_in, unsigned char * image_out, int width, int height, unsigned int *cdf, unsigned int *cdfmin) {
    unsigned int imageSize = width * height;
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    //Equalize
    if (x < width && y < height){
        image_out[(y*width + x)] = scale(cdf[image_in[y*width + x]], *cdfmin, imageSize);
    }
}

void printHistogram(unsigned int* histogram) {
    printf("[");
    for (int i=0; i<GRAYLEVELS; i++) {
        if (i == GRAYLEVELS-1){
            printf("%d]\n", histogram[i]);
            return;
        }   
        printf("%d, ", histogram[i]);
    }
}

unsigned char Scale(unsigned int cdf, unsigned int cdfmin, unsigned int imageSize){
    float scale;
    scale = (float)(cdf - cdfmin) / (float)(imageSize - cdfmin);
    scale = round(scale * (float)(GRAYLEVELS-1));
    return (int)scale;
}
void Equalize(unsigned char *image_in, unsigned char *image_out, int width, int height, unsigned int *cdf){
    unsigned int imageSize = width * height;
    unsigned int cdfmin = 2; 
    //Equalize: namig: blok niti naj si CDF naloži v skupni pomnilnik
    for (int i=0; i<height; i++) {
        for (int j=0; j<width; j++) {
            image_out[(i*width + j)] = Scale(cdf[image_in[i*width + j]], cdfmin, imageSize);
        }
    }
}

int main(int argc, char *argv[]) {
    
    char imageInName[255]; char imageoutNamePNG[255]; char imageoutNameJPG[255]; char imageInFormat[5];
    snprintf(imageInName, 255, "%s", argv[1]);
    snprintf(imageInFormat, 5, "%s", argv[2]); 
    strncat(imageInName, imageInFormat, 4);
    snprintf(imageoutNamePNG, 255, "%s", argv[1]); snprintf(imageoutNameJPG, 255, "%s", argv[1]);
    strncat(imageoutNamePNG, "_out.png", 12); strncat(imageoutNameJPG, "_out.jpg", 12);

    // Read image from file
    int width, height, cpp;
    // read only DESIRED_NCHANNELS channels from the input image:
    unsigned char *h_imageIn = stbi_load(imageInName, &width, &height, &cpp, DESIRED_NCHANNELS);
    if(h_imageIn == NULL) {
        printf("Error in loading the image\n");
        return 1;
    }
    printf("Loaded image W = %d, H = %d, actual cpp = %d \n", width, height, cpp);
	
    //Allocate memory for raw output image data, histogram, and CDF 
    unsigned char *h_imageOut = (unsigned char *)malloc(width * height * sizeof(unsigned char));
	unsigned int *h_histogram = (unsigned int *)malloc(GRAYLEVELS * sizeof(unsigned int));
    unsigned int *h_cdf = (unsigned int *)malloc(GRAYLEVELS * sizeof(unsigned int));
    unsigned int *h_cdfmin = (unsigned int *)malloc(sizeof(unsigned int));
    

    dim3 blockSize(16, 16);
    dim3 gridSize(ceil((float) width / blockSize.x), ceil((float) height / blockSize.y));
    unsigned int *d_histogram;
    unsigned char *d_imageIn;
    unsigned char *d_imageOut;
    unsigned int *d_cdf;
    unsigned int *d_cdfmin;
    hipMalloc(&d_histogram, GRAYLEVELS * sizeof(unsigned int));
    hipMalloc(&d_imageIn, width * height * sizeof(unsigned char));
    hipMalloc(&d_imageOut, width * height * sizeof(unsigned char));
    hipMalloc(&d_cdf, GRAYLEVELS * sizeof(unsigned int));
    hipMalloc(&d_cdfmin, sizeof(unsigned int));

    //timestart
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    float milliseconds = 0;

	// Histogram equalization steps: 
	// 1. Create the histogram for the input grayscale image.
    hipMemcpy(d_imageIn, h_imageIn, width * height * sizeof(unsigned char), hipMemcpyHostToDevice);
    
    for (int n = 0; n < N; n++){

        CalculateHistogramKernel<<<gridSize, blockSize>>>(d_imageIn, width, height, d_histogram);
        #ifdef PRINT_HISTOGRAMS
            hipMemcpy(h_histogram, d_histogram, GRAYLEVELS * sizeof(unsigned int), hipMemcpyDeviceToHost);
            printf("INITIAL HISTOGRAM \n");
            printHistogram(h_histogram);
            printf("\n");
        #endif  // PRINT_HISTOGRAMS

	    //  2. Calculate the cumulative distribution histogram.
        #ifdef CDF_NAIVE
        CalculateCDF_naive<<<1, 256>>>(d_histogram, d_cdf);
        #endif  // CDF_NAIVE

        #ifdef CDF_WE
        CalculateCDF_we<<<1, 128>>>(d_histogram, d_cdf);
        #endif  // CDF_WE

        #ifdef CDF_WE_MBCF
        CalculateCDF_we_mbcf<<<1, 128>>>(d_histogram, d_cdf);
        #endif  // CDF_WE_MBCF
    
	    //  3. Calculate the new gray-level values through the general histogram equalization formula and assign new pixel values
        findMinKernel<<<1, 128>>>(d_cdf, d_cdfmin);
        hipMemcpy(h_cdfmin, d_cdfmin, sizeof(unsigned int), hipMemcpyDeviceToHost);
        //printf("CDFMIN: %d\n", *h_cdfmin);
        //hipMemcpy(h_cdf, d_cdf, GRAYLEVELS * sizeof(unsigned int), hipMemcpyDeviceToHost);

        EqualizeKernel<<<gridSize, blockSize>>>(d_imageIn, d_imageOut, width, height, d_cdf, d_cdfmin);
        //Equalize(h_imageIn, h_imageOut, width, height, h_cdf);
        hipMemcpy(h_imageOut, d_imageOut, height * width * sizeof(unsigned char), hipMemcpyDeviceToHost);
    
        #ifdef PRINT_HISTOGRAMS
            printf("CDF\n");
            hipMemcpy(h_cdf, d_cdf, GRAYLEVELS * sizeof(unsigned int), hipMemcpyDeviceToHost);
            printHistogram(h_cdf);
            printf("\n");
            CalculateHistogramKernel<<<gridSize, blockSize>>>(d_imageOut, width, height, d_histogram);
            hipMemcpy(h_histogram, d_histogram, GRAYLEVELS * sizeof(unsigned int), hipMemcpyDeviceToHost);
            printf("EQUALIZED HISTOGRAM \n");
            printHistogram(h_histogram);
        #endif  // PRINT_HISTOGRAMS
    }

    //time stop
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Kernel Execution time is: %0.4f milliseconds \n", milliseconds/N);


    //Free cuda memory
    hipFree(d_histogram);
    hipFree(d_imageIn);
    hipFree(d_imageOut);
    hipFree(d_cdf);
    hipFree(d_cdfmin);

    // write output image:
    stbi_write_png(imageoutNamePNG, width, height, DESIRED_NCHANNELS, h_imageOut, width * DESIRED_NCHANNELS);
    stbi_write_jpg(imageoutNameJPG, width, height, DESIRED_NCHANNELS, h_imageOut, 100);
    //stbi_write_jpg("out.jpg", width, height, DESIRED_NCHANNELS, h_imageOut, 100);

	//Free memory
	free(h_imageIn);
    free(h_imageOut);
	free(h_histogram);
    free(h_cdf);

    

	return 0;
}

